#include "hip/hip_runtime.h"
#include <stdio.h>

#define SYSTEM_SIZE {{ SYSTEM_SIZE }}

typedef {{ Real }} Real;

/**
 * Computes the value of the threadId-th component of the function
 * F(t) = (f1(t), ..., fn(t)) and stores it in the memory pointed by f
 * @param Real  t  Value of the time in which the system is solved
 * @param Real* y  Initial conditions for the system: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 * @param Real* f  Computed value of the function: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 */
__device__ void computeComponent(int threadId, Real x, Real* y, Real* f){
    // Jinja template that renders to a switch in which every thread computes
    // a different equation and stores it in the corresponding position in f
    switch(threadId) {
        {% for i, function in SYSTEM_FUNCTIONS %}
            case {{ i }}:
                f[threadId] = {{ function }};
                break;
        {% endfor %}
    }
    printf("ThreadId %d - COMPU: %.5f, %.5f, %.5f, %5f\n", threadId, x, y[0], y[1], f[threadId]);
}

/**
 * Computes a step of the Runge Kutta 4 algorithm, storing the results in the
 * GPU array pointed by devInitCond.
 * @param {[type]} Real x0           Value of the time in which the system is
 * solved
 * @param {[type]} void  *devInitCond Pointer to a GPU array with the initial
 * conditions, also used as output for the evolution of the system.
 * @param {[type]} Real dx           Step size.
 * @param {[type]} Real tolerance    Error tolerance, used in the adaptative
 *                      step size computation.
 * @return Real The new step size.
 */
__global__ void RK4Solve(void* devX0, void *devInitCond, void* devStep, Real tolerance){
    // Retrieve the identifiers of the thread in the block and of the block in
    // the grid
    int threadId = threadIdx.x + threadIdx.y * blockDim.x;
    int blockId =  blockIdx.x  + blockIdx.y  * gridDim.x;

    // Assure the running thread is a useful thread :)
    if(threadId < SYSTEM_SIZE){
        // Arrays to store fourth and fifth order solutions.
        __shared__ Real rk4[SYSTEM_SIZE], rk5[SYSTEM_SIZE];

        // First try of the step size
        Real* globalStep = (Real*)devStep;
        Real dx = *globalStep;

        // Time
        Real* globalX0 = (Real*)devX0;
        Real x0 = *globalX0;

        // Retrieve the initial conditions this block will work with
        Real* globalInitCond = (Real*)devInitCond + blockId*SYSTEM_SIZE;

        // Get the initial condition this thread will work with
        Real y0 = globalInitCond[threadId];

        // Auxiliar computation arrays
        __shared__ Real k1[SYSTEM_SIZE],
                        k2[SYSTEM_SIZE],
                        k3[SYSTEM_SIZE],
                        k4[SYSTEM_SIZE],
                        k5[SYSTEM_SIZE],
                        k6[SYSTEM_SIZE];

        // New value of the system
        __shared__ Real y1[SYSTEM_SIZE];

        // Local errors
        __shared__ Real errors[SYSTEM_SIZE];
        Real delta, R = 0.0;
        Real err;

        do{
            // K1 computation
            y1[threadId] = y0;
            __syncthreads();
            computeComponent(threadId, x0, y1, k1);
            __syncthreads();

            // K2 computation
            y1[threadId] = y0 + dx*(1./4.)*k1[threadId];
            __syncthreads();
            computeComponent(threadId, x0 + (1./4.)*dx, y1, k2);
            __syncthreads();

            // K3 computation
            y1[threadId] = y0 + dx*((3./32.)*k1[threadId] +
                                    (9./32.)*k2[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (3./8.)*dx, y1, k3);
            __syncthreads();

            // K4 computation
            y1[threadId] = y0 + dx*(  (1932./2197.)*k1[threadId]
                                    - (7200./2197.)*k2[threadId]
                                    + (7296./2197.)*k3[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (12./13.)*dx, y1, k4);
            __syncthreads();

            // K5 computation
            y1[threadId] = y0 + dx*( (439./216.)*k1[threadId]
                                    - 8.*k2[threadId]
                                    + (3680./513.)*k3[threadId]
                                    - (845./4104.)*k4[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + dx, y1, k5);
            __syncthreads();

            // K6 computation
            y1[threadId] = y0 + dx*(-(8./27.)*k1[threadId]
                                    + 2.*k2[threadId]
                                    - (3544./2565.)*k3[threadId]
                                    + (1859./4104.)*k4[threadId]
                                    - (11./40.)*k5[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (1/2)*dx, y1, k6);
            __syncthreads();

            // Compute fourth and fifth order solutions
            rk4[threadId] = y0 + dx*( (25./216.)*k1[threadId]
                                    + (1408./2565.)*k3[threadId]
                                    + (2197./4101.)*k4[threadId]
                                    - (1./5.)*k5[threadId]);

            rk5[threadId] = y0 + dx*( (16./135.)*k1[threadId]
                                    + (6656./12825.)*k3[threadId]
                                    + (28561./56430.)*k4[threadId]
                                    - (9./50.)*k5[threadId]
                                    + (2./55.)*k6[threadId]);

            // Real sc = tolerance*(1 + fmax(y0, rk5[threadId]));
            //
            // // Retrieve the local errors
            // Real quotient = (rk5[threadId] - rk4[threadId])/tolerance;
            // errors[threadId] = quotient*quotient;
            // __syncthreads();
            //
            // printf("ThreadId %d - QUOTI: %.20f, %.20f\n", threadId, quotient, tolerance);
            //
            // printf("ThreadId %d - K1234: K1:%.7f, K2:%.7f, K3:%.7f, K4:%.7f, K5:%.7f, K6:%.7f\n", threadId, k1[threadId], k2[threadId], k3[threadId], k4[threadId], k5[threadId], k6[threadId]);
            // printf("ThreadId %d - RK4 5: %.20f, %.20f\n", threadId, rk4[threadId], rk5[threadId]);
            // printf("ThreadId %d - ERROR: %.20f\n", threadId, errors[threadId]);
            //
            // // Compute the distance between both solutions with the usual
            // // reduction technique, storing it in errors[0]. Note that the
            // // number of threads has to be a power of 2.
            // for(int s=(blockDim.x*blockDim.y)/2; s>0; s>>=1){
            //     if (threadId < s) {
            //         printf("ThreadId %d - SUMMS: S: %d, error: (%.10f, %.10f)\n", threadId, s, errors[threadId], errors[threadId+s]);
            //         errors[threadId] = errors[threadId] + errors[threadId + s];
            //     }
            //
            //     __syncthreads();
            // }
            //
            //
            // if(threadId == 0)
            //     printf("ThreadId %d - SUMMS: GLOBAL ERROR: %.20f\n", threadId, errors[0]);
            //
            // err = sqrt(errors[0]/SYSTEM_SIZE);
            //
            // #define FACMAX 1.5
            // #define FACMIN 0.1
            // #define FAC 0.8
            // dx *= fmin(FACMAX, fmax(FACMIN, FAC*pow(1./err, 0.2)));

            // // Update the step
            // R = sqrt(errors[0])/dx;
            // if(R > tolerance){
            //     delta = pow((Real)0.84*(tolerance/R), (Real)0.25);
            //     dx *= delta;
            // }

            err = 0.;

            if(threadId == 0){
                if(err > 1.){
                    printf("\n###### CHANGE: err: %.20f, dx: %.20f\n\n", err, dx);
                }
                else{
                    printf("\n###### ======:  err: %.20f, dx: %.20f\n\n", err, dx);
                }
            }
        }while(err > 1.);

        // Update system value in the global memory.
        globalInitCond[threadId] = rk5[threadId];

        // Update global step and time. Do it just once.
        if(threadId == 0){
            *globalStep = dx;
            *globalX0 += dx;
        }

    } // If threadId < SYSTEM_SIZE
}
