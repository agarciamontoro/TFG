#include "hip/hip_runtime.h"
// PABLO: It's important that we comment the code much more, explaining the steps of the algorithm
// in a comprehensive way because this is meant to be read and modified by poor guys in science that does not
// have to full understand ARK4. This does not need to be a brief text-book but a lot more information regarding
// each step (and more important: What is the porupose of each variable in the algorithm) truly helps to understand how
// the algorithm works and how the implementation works. And this will make this very pleasent to read in the future.



#include <stdio.h>

#define SYSTEM_SIZE {{ SYSTEM_SIZE }}
{{ DEBUG }}

typedef {{ Real }} Real;

/**
 * Computes the value of the threadId-th component of the function
 * F(t) = (f1(t), ..., fn(t)) and stores it in the memory pointed by f
 * @param Real  t  Value of the time in which the system is solved
 * @param Real* y  Initial conditions for the system: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 * @param Real* f  Computed value of the function: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 */
__device__ void computeComponent(int threadId, Real x, Real* y, Real* f){
    // Jinja template that renders to a switch in which every thread computes
    // a different equation and stores it in the corresponding position in f
    switch(threadId) {
        {% for i, function in SYSTEM_FUNCTIONS %}
            case {{ i }}:
                f[threadId] = {{ function }};
                break;
        {% endfor %}
    }
}

/**
 * Computes a step of the Runge Kutta 4 algorithm, storing the results in the
 * GPU array pointed by devInitCond.
 * @param {[type]} Real x0           Value of the time in which the system is
 * solved
 * @param {[type]} void  *devInitCond Pointer to a GPU array with the initial
 * conditions, also used as output for the evolution of the system.
 * @param {[type]} Real h           Step size.
 * @param {[type]} Real tolerance    Error tolerance, used in the adaptative
 *                      step size computation.
 * @return Real The new step size.
 */
 __global__ void RK4Solve(void* devX0, Real xend, void *devInitCond, Real h,
                          Real hmax, void* globalRtoler, void* globalAtoler, Real safe, Real fac1, Real fac2, Real beta,
                          Real uround){

    // Retrieve the ids of the thread in the block and of the block in the grid
    int threadId = threadIdx.x + threadIdx.y * blockDim.x;
    int blockId =  blockIdx.x  + blockIdx.y  * gridDim.x;

    #ifdef DEBUG
        printf("ThreadId %d - INITS: x0=%.20f, xend=%.20f, y0=(%.20f, %.20f)\n", threadId, *((Real*)devX0), xend, ((Real*)devInitCond)[0], ((Real*)devInitCond)[1]);
    #endif

    // Assure the running thread is a useful thread :)
    if(threadId < SYSTEM_SIZE){
        // Arrays to store intermediate solutions.
        __shared__ Real solution[SYSTEM_SIZE];

        // First try of the step size
        // TODO: Implement the hinit method
        Real hnew;

        // Time
        Real* globalX0 = (Real*)devX0;
        Real x0 = *globalX0;

        // Retrieve the initial conditions this block will work with
        Real* globalInitCond = (Real*)devInitCond + blockId*SYSTEM_SIZE;

        // Get the initial condition this thread will work with
        Real y0 = globalInitCond[threadId];

        // Auxiliar computation arrays
        __shared__ Real k1[SYSTEM_SIZE],
                        k2[SYSTEM_SIZE],
                        k3[SYSTEM_SIZE],
                        k4[SYSTEM_SIZE],
                        k5[SYSTEM_SIZE],
                        k6[SYSTEM_SIZE],
                        k7[SYSTEM_SIZE];
        __shared__ Real y1[SYSTEM_SIZE];

        // Local errors
        __shared__ Real errors[SYSTEM_SIZE];
        Real err;

        // Initial values for the loop variables
        // PABLO: Add comment explaining what is what and what is its pourpose.
        Real facold = 1.0E-4;
        Real expo1 = 0.2 - beta * 0.75;
        Real facc1 = 1.0 / fac1;
        Real facc2 = 1.0 / fac2;

        // PABLO: Explain what Error tolerances are and why do you need each one, here or some place else.
        // Error tolerances
        Real* atoler = (Real*) globalAtoler;
        Real* rtoler = (Real*) globalRtoler;
        Real atoli = atoler[threadId];
        Real rtoli = rtoler[threadId];

        // PABLO: EXPLAIN STUFF :)
        // More stuff
        bool last  = false;
        Real fac11, fac;
        Real sqr;

        Real reject = false;

        // PABLO: When starting a loop like this, preprend a comment (middle-long extension) explain
        // what the loop is going to do and what is the WHILE condition that will follows. This improoves
        // readability
        do{
            // TODO: Check that the step size is not too small

            if ((x0 + 1.01*h - xend) > 0.0){
              h = xend - x0;
              last = true;
            }

            // K1 computation
            y1[threadId] = y0;
            __syncthreads();
            computeComponent(threadId, x0, y1, k1);
            __syncthreads();

            // K2 computation
            y1[threadId] = y0 + h*(1./5.)*k1[threadId];
            __syncthreads();
            computeComponent(threadId, x0 + (1./5.)*h, y1, k2);
            __syncthreads();

            // K3 computation
            y1[threadId] = y0 + h*((3./40.)*k1[threadId] +
                                    (9./40.)*k2[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (3./10.)*h, y1, k3);
            __syncthreads();

            // K4 computation
            y1[threadId] = y0 + h*(  (44./45.)*k1[threadId]
                                    - (56./15.)*k2[threadId]
                                    + (32./9.)*k3[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (4./5.)*h, y1, k4);
            __syncthreads();

            // K5 computation
            y1[threadId] = y0 + h*( (19372./6561.)*k1[threadId]
                                    - (25360./2187.)*k2[threadId]
                                    + (64448./6561.)*k3[threadId]
                                    - (212./729.)*k4[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + (8./9.)*h, y1, k5);
            __syncthreads();

            // K6 computation
            y1[threadId] = y0 + h*((9017./3168.)*k1[threadId]
                                    - (355./33.)*k2[threadId]
                                    + (46732./5247.)*k3[threadId]
                                    + (49./176.)*k4[threadId]
                                    - (5103./18656.)*k5[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + h, y1, k6);
            __syncthreads();

            // K7 computation. Maybe store it in K2 and get rid of K7? :D
            y1[threadId] = y0 + h*((35./384.)*k1[threadId]
                                    + (500./1113.)*k3[threadId]
                                    + (125./192.)*k4[threadId]
                                    - (2187./6784.)*k5[threadId]
                                    + (11./84.)*k6[threadId]);
            __syncthreads();
            computeComponent(threadId, x0 + h, y1, k7);
            __syncthreads();

            // Compute solution and local error
            // PABLO: More info about local error, which technique is being used, how? Link to the Butcher table...etc.
            // We do not need a text book about the algorithm but further explanation for the novice is important here.
            solution[threadId] = y1[threadId];
            errors[threadId] = h*((71./57600.)*k1[threadId]
                                - (71./16695.)*k3[threadId]
                                + (71./1920.)*k4[threadId]
                                - (17253./339200.)*k5[threadId]
                                + (22./525.)*k6[threadId]
                                - (1./40.)*k7[threadId]);

            #ifdef DEBUG
                printf("ThreadId %d - K 1-7: K1:%.20f, K2:%.20f, K3:%.20f, K4:%.20f, K5:%.20f, K6:%.20f, K7:%.20f\n", threadId, k1[threadId], k2[threadId], k3[threadId], k4[threadId], k5[threadId], k6[threadId], k7[threadId]);
                printf("ThreadId %d - Local: sol: %.20f, error: %.20f\n", threadId, solution[threadId], errors[threadId]);
            #endif

            // Compute scale factor
            // PABLO: Explain the scale factor
            Real sk = atoli + rtoli*fmax(abs(y0), abs(solution[threadId]));

            // Compute the summands of the total error
            // PABLO: Explain "summands of the total error"
            sqr = (errors[threadId])/sk;
            errors[threadId] = sqr*sqr;
            __syncthreads();

            #ifdef DEBUG
                printf("ThreadId %d - Diffs: sqr: %.20f, sk: %.20f\n", threadId, sqr, sk);
            #endif

            //PABLO: Explain this technique in detail here. Notice that if someone (i.e. you or me)
            // in the future need to change this and completely forgots (or do not know) how the parallel
            // reduce works this will not be understanded. Also notice that this needs to be further explained
            // because of the power of 2 restriction.

            // Add the local errors with the usual reduction technique, storing
            // it in errors[0]. Note that the number of threads has to be a
            // power of 2.
            for(int s=(blockDim.x*blockDim.y)/2; s>0; s>>=1){
                if (threadId < s) {
                    errors[threadId] = errors[threadId] + errors[threadId + s];
                }

                __syncthreads();
            }

            // Compute the total error
            err = sqrt(errors[0]/(Real)SYSTEM_SIZE);

            // Explain this steps in more detail. A brief comment paragraph about what is going to happend and why.
            /* computation of hnew */
            fac11 = pow (err, expo1);
            /* Lund-stabilization */
            fac = fac11 / pow(facold,beta);
            /* we require fac1 <= hnew/h <= fac2 */
            fac = fmax(facc2, fmin(facc1, fac/safe));
            hnew = h / fac;

            #ifdef DEBUG
                printf("ThreadId %d - H aux: expo1: %.20f, err: %.20f, fac11: %.20f, facold: %.20f, fac: %.20f\n", threadId, expo1, err, fac11, facold, fac);
                printf("ThreadId %d - H new: prevH: %.20f, newH: %.20f\n", threadId, hnew);
            #endif

            // STEP REJECTED
            if( err > 1.){
                hnew = h / fmin(facc1, fac11/safe);
                reject = true;
            }
            // STEP ACCEPTED
            else{
                // TODO: Stiffness detection

                facold = fmax(err, 1.0e-4);
                x0 += h;

                if (hnew > hmax)
                    hnew = hmax;

                if (reject)
                    hnew = fmin(fabs(hnew), fabs(h));

                y0 = solution[threadId];

                reject = false;
            }

            h = hnew;

            #ifdef DEBUG
                if(threadId == 0){
                    if(err > 1.){
                        printf("\n###### CHANGE: err: %.20f, h: %.20f\n\n", err, h);
                    }
                    else{
                        printf("\n###### ======:  err: %.20f, h: %.20f\n\n", err, h);
                    }
                }
            #endif
        }while(!last);

        // Update system value in the global memory.
        globalInitCond[threadId] = solution[threadId];

        // Update global step and time. Do it just once.
        if(threadId == 0){
            *globalX0 = x0;
        }

    } // If threadId < SYSTEM_SIZE
}
