#include "hip/hip_runtime.h"
#define SYSTEM_SIZE {{ SYSTEM_SIZE }}

typedef {{ Real }} Real;

/**
 * Returns the block identifier in a 2D grid
 * @return int Block identifier in which the running thread resides
 */
__device__ inline int getBlockId(){
   return blockIdx.x + blockIdx.y * gridDim.x;
}

/**
 * Returns the thread identifier in a 2D block
 * @return int Thread identifier local to the block
 */
__device__ inline int getThreadId(){
    return (threadIdx.y * blockDim.x) + threadIdx.x;
}

/**
 * Computes the value of the threadId-th component of the function
 * F(t) = (f1(t), ..., fn(t)) and stores it in the memory pointed by f
 * @param Real  t  Value of the time in which the system is solved
 * @param Real* y  Initial conditions for the system: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 * @param Real* f  Computed value of the function: a vector whose lenght
 *                  shall be the same as the number of equations in the system
 */
__device__ void computeComponent(int threadId, Real x, Real* y, Real* f){
    // Jinja template that renders to a switch in which every thread computes
    // a different equation and stores it in the corresponding position in f
    switch(threadId) {
        {% for i, function in SYSTEM_FUNCTIONS %}
            case {{ i }}:
                f[threadId] = {{ function }};
                break;
        {% endfor %}
    }
}

/**
 * Computes a step of the Runge Kutta 4 algorithm, storing the results in the
 * GPU array pointed by devInitCond.
 * @param {[type]} Real x0           Value of the time in which the system is
 * solved
 * @param {[type]} void  *devInitCond Pointer to a GPU array with the initial
 * conditions, also used as output for the evolution of the system.
 * @param {[type]} Real dx           Step size.
 */
__global__ void RK4Solve(Real x0, void *devInitCond, Real dx){
    // Retrieve the identifiers of the thread in the block and of the block in
    // the grid
    int threadId = getThreadId();
    int blockId = getBlockId();

    // Retrieve the initial conditions this block will work with
    Real* globalInitCond = (Real*)devInitCond + blockId*SYSTEM_SIZE;

    // Copy the initial conditions to shared memory (as there are as many
    // initial conditions as equations on the system, each thread can copy one
    // of them).
    __shared__ Real y0[SYSTEM_SIZE];
    y0[threadId] = globalInitCond[threadId];

    __syncthreads();

    // Auxiliar computation arrays
    __shared__ Real k1[SYSTEM_SIZE],
                     k2[SYSTEM_SIZE],
                     k3[SYSTEM_SIZE],
                     k4[SYSTEM_SIZE];

    // New value of the system
    __shared__ Real y1[SYSTEM_SIZE];

    // K1 computation
    computeComponent(threadId, x0, y0, k1);
    __syncthreads();

    // K2 computation
    y1[threadId] = y0[threadId] + 0.5*dx*k1[threadId];
    __syncthreads();

    computeComponent(threadId, x0 + 0.5*dx, y1, k2);
    __syncthreads();

    // K3 computation
    y1[threadId] = y0[threadId] + 0.5*dx*k2[threadId];
    __syncthreads();

    computeComponent(threadId, x0 + 0.5*dx, y1, k3);
    __syncthreads();

    // K4 computation
    y1[threadId] = y0[threadId] + dx*k3[threadId];
    __syncthreads();

    computeComponent(threadId, x0 + dx, y1, k4);
    __syncthreads();

    // Update system value in the global memory
    globalInitCond[threadId] += dx*(k1[threadId] +
                                    2*(k2[threadId]+k3[threadId]) +
                                    k4[threadId])/6;
}
