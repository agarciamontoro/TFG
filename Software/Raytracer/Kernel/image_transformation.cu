#include "hip/hip_runtime.h"
#ifndef __IMAGE_TRANSFORMATION__
#define __IMAGE_TRANSFORMATION__

#include "Raytracer/Kernel/common.cu"

#define TXT_COLS 2363
#define TXT_ROWS 500

__global__ void generate_image(void* devRayCoordinates, void* devStatus,
                               void* devDiskTexture,
                               int diskRows, int diskCols,
                               void* devSphereTexture,
                               int sphereRows, int sphereCols,
                               void* devImage){
    // Compute pixel's row and col of this thread
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < IMG_ROWS && col < IMG_COLS){
        // Compute pixel unique identifier
        int pixel = row*IMG_COLS + col;

        // Retrieve status of the current pixel
        int* globalStatus = (int*) devStatus;
        globalStatus += pixel;
        int status = *globalStatus;

        // Locate the coordinates of the current ray
        Real* globalRaycoords = (Real*) devRayCoordinates;
        globalRaycoords += pixel * SYSTEM_SIZE;

        // Retrieve image and texture pointers
        Real* diskTexture = (Real*) devDiskTexture;
        Real* sphereTexture = (Real*) devSphereTexture;
        Real* image = (Real*) devImage;

        // Locate the image pixel that corresponds to this thread
        image += pixel * 3;

        // Copy the coordinates of the current ray to local memory
        // FIXME: Is this efficient? We only access once to the memory.
        Real rayCoords[SYSTEM_SIZE];
        memcpy(rayCoords, globalRaycoords, sizeof(Real)*SYSTEM_SIZE);

        // Variables to hold the ray coordinates
        Real r, theta, phi;

        r = rayCoords[0];
        theta = rayCoords[1];
        phi = rayCoords[2];

        // Variables to hold the texel coordinates
        Real x, y, z;
        int u, v, texel;

        Real rNormalized;

        switch(status){
            case DISK:
                rNormalized = (r - innerDiskRadius) / (outerDiskRadius - innerDiskRadius);

                u = round((sin(phi) + 1)/2 * diskCols);
                v = round(rNormalized * diskRows);

                texel = v*diskCols + u;
                diskTexture += texel * 3;

                memcpy(image, diskTexture, 3*sizeof(Real));

                break;

            case SPHERE:
                // x = sin(theta) * cos(phi);
                // y = sin(theta) * sin(phi);
                // z = cos(theta);
                //
                // u = round((0.5 + atan2(z, x) / (2*Pi)) * sphereCols);
                // v = round((0.5 - asin(y) / Pi) * sphereRows);

                u = round(sphereCols * phi / (2*Pi));
                v = round(sphereRows * theta / Pi);

                texel = v*sphereCols + u;
                sphereTexture += texel * 3;

                memcpy(image, sphereTexture, 3*sizeof(Real));
                break;

            case HORIZON:
                image[0] = 0;
                image[1] = 0;
                image[2] = 0;
                break;
        }
    }

}

#endif // __IMAGE_TRANSFORMATION__
