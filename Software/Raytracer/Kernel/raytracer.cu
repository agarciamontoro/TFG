#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include "Raytracer/Kernel/common.cu"

#define Pi M_PI
#define SYSTEM_SIZE 5

__device__ void getCanonicalMomenta(Real rayTheta, Real rayPhi, Real* pR,
                                    Real* pTheta, Real* pPhi){
    // **************************** SET NORMAL **************************** //
    // Cartesian components of the unit vector N pointing in the direction of
    // the incoming ray
    Real Nx = sin(rayTheta) * cos(rayPhi);
    Real Ny = sin(rayTheta) * sin(rayPhi);
    Real Nz = cos(rayTheta);

    // ********************** SET DIRECTION OF MOTION ********************** //
    // Compute denominator, common to all the cartesian components
    Real den = 1. - __camBeta * Ny;

    // Compute factor common to nx and nz
    Real fac = -sqrt(1. - __camBeta*__camBeta);

    // Compute cartesian coordinates of the direction of motion. See(A.9)
    Real nY = (-Ny + __camBeta) / den;
    Real nX = fac * Nx / den;
    Real nZ = fac * Nz / den;

    // Convert the direction of motion to the FIDO's spherical orthonormal
    // basis. See (A.10)
    Real nR = nX;
    Real nTheta = -nZ;
    Real nPhi = nY;

    // *********************** SET CANONICAL MOMENTA *********************** //
    // Compute energy as measured by the FIDO. See (A.11)
    Real E = 1. / (__alpha + __omega * __pomega * nPhi);

    // Set conserved energy to unity. See (A.11)
    // Real pt = -1;

    // Compute the canonical momenta. See (A.11)
    *pR = E * __ro * nR / sqrt(__delta);
    *pTheta = E * __ro * nTheta;
    *pPhi = E * __pomega * nPhi;
}

__device__ void getConservedQuantities(Real pTheta, Real pPhi, Real* b,
                                       Real* q){
    // ********************* GET CONSERVED QUANTITIES ********************* //
    // Get conserved quantities. See (A.12)
    *b = pPhi;

    Real sinT = sin(__camTheta);
    Real sinT2 = sinT*sinT;

    Real cosT = cos(__camTheta);
    Real cosT2 = cosT*cosT;

    Real pTheta2 = pTheta*pTheta;
    Real b2 = pPhi*pPhi;

    *q = pTheta2 + cosT2*((b2/sinT2) - __a2);
}

__global__ void setInitialConditions(void* devInitCond,void* devConstants,
                                     Real pixelWidth, Real pixelHeight){
    // Unique identifier of this thread
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute pixel's row and col of this thread
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if(row < IMG_ROWS && col < IMG_COLS){
        // Compute pixel unique identifier for this thread
        int pixel = row*IMG_COLS + col;

        // Pointer for the initial conditions of this ray (block)
        Real* globalInitCond = (Real*) devInitCond;
        Real* initCond = globalInitCond + pixel*SYSTEM_SIZE;

        // Pointer for the constants of this ray (block)
        Real* globalConstants = (Real*) devConstants;
        Real* constants = globalConstants + pixel*2;

        // Compute pixel position in the physical space
        Real x = - (col + 0.5 - IMG_COLS/2) * pixelWidth;
        Real y = (row + 0.5 - IMG_ROWS/2) * pixelHeight;

        // Compute direction of the incoming ray in the camera's reference
        // frame
        Real rayPhi = Pi + atan(x / __d);
        Real rayTheta = Pi/2 + atan(y / sqrt(__d*__d + x*x));

        // Compute canonical momenta of the ray and the conserved quantites b
        // and q
        Real pR, pTheta, pPhi, b, q;
        getCanonicalMomenta(rayTheta, rayPhi, &pR, &pTheta, &pPhi);
        getConservedQuantities(pTheta, pPhi, &b, &q);

        #ifdef DEBUG
            if(blockIdx.x == 0 && blockIdx.y == 0){
                printf("%.20f, %.20f\n", x, y);
                printf("INICIALES: theta = %.20f, phi = %.20f, pR = %.20f, pTheta = %.20f, pPhi = %.20f, b = %.20f, q = %.20f", rayTheta, rayPhi, pR, pTheta, pPhi, b, q);
            }
        #endif

        // Save ray's initial conditions
        initCond[0] = __camR;
        initCond[1] = __camTheta;
        initCond[2] = __camPhi;
        initCond[3] = pR;
        initCond[4] = pTheta;

        // Save ray's constants
        constants[0] = b;
        constants[1] = q;
    }
}

__device__ int detectCollisions(Real prevThetaCentered,
                                Real currentThetaCentered, Real prevR,
                                Real currentR){
    if (currentR <= horizonRadius){
        return HORIZON;
    }

    if(prevThetaCentered*currentThetaCentered < 0 &&
       prevR > innerDiskRadius && currentR > innerDiskRadius &&
       prevR < outerDiskRadius && currentR < outerDiskRadius){
        return DISK;
    }

    return SPHERE;
}



#include "Raytracer/Kernel/solver.cu"

__global__ void kernel(Real x0, Real xend, void* devInitCond, Real h,
                       Real hmax, void* devData, int dataSize,
                       void* devStatus, Real resolution){
   // Compute pixel's row and col of this thread
   int row = blockDim.y * blockIdx.y + threadIdx.y;
   int col = blockDim.x * blockIdx.x + threadIdx.x;

   if(row < IMG_ROWS && col < IMG_COLS){
       // Compute pixel unique identifier for this thread
       int pixel = row*IMG_COLS + col;

       // Array of status flags: at the output, the (x,y)-th element will be 0
       // if any error ocurred (namely, the step size was made too small) and
       // 1 if the computation succeded
       int* globalStatus = (int*) devStatus;
       globalStatus += pixel;
       int status = *globalStatus;

       // Retrieve the position where the initial conditions this block will
       // work with are.
       // Each block, absolutely identified in the grid by blockId, works with
       // only one initial condition (that has N elements, as N equations are
       // in the system). Then, the position of where these initial conditions
       // are stored in the serialized vector can be computed as blockId * N.
       Real* globalInitCond = (Real*) devInitCond;
       globalInitCond += pixel * SYSTEM_SIZE;

       // Pointer to the additional data array used by computeComponent
       Real* globalData = (Real*) devData;
       globalData += pixel * dataSize;

       // Shared arrays to store the initial conditions and the additional
       // data
       Real initCond[SYSTEM_SIZE], data[DATA_SIZE];

       for(int i = 0; i < SYSTEM_SIZE; i++){
           initCond[i] = globalInitCond[i];
       }

       for(int i = 0; i < DATA_SIZE; i++){
           data[i] = globalData[i];
       }

       // Initialize previous theta and r to the initial conditions
       Real prevThetaCentered, prevR, currentThetaCentered, currentR;

       prevR = initCond[0];
       prevThetaCentered = initCond[1] - HALF_PI;

       // Local variable to know the status of the ray

       // Current time
       Real x = x0;
       SolverStatus solverStatus;

       while(status == SPHERE && x > xend){
           solverStatus = RK4Solve(x, x + resolution, initCond, &h, resolution, data);

           if(solverStatus == RK45_SUCCESS){
               currentR = initCond[0];
               currentThetaCentered = initCond[1] - HALF_PI;

               status = detectCollisions(prevThetaCentered,
                                         currentThetaCentered,
                                         prevR, currentR);

               if(status == DISK){
                   bisect(initCond, data, h);
               }
           }
           else{
               status = HORIZON;
           }

           prevR = currentR;
           prevThetaCentered = currentThetaCentered;

           x += resolution;

       } // While globalStatus == SPHERE and x > xend


       *globalStatus = status;

       for(int i = 0; i < SYSTEM_SIZE; i++){
           globalInitCond[i] = initCond[i];
       }

   } // If threadId < NUM_PIXELS
}
