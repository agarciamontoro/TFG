#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include "Raytracer/numericalMethods.cu"
#include "Raytracer/definitions.c"
#include "RK4/rk4_kernel_adapted.cu"

#define SYSTEM_SIZE {{ SYSTEM_SIZE }}
{{ DEBUG }}

#define J I
#define Pi M_PI

typedef enum origin{
    HORIZON,
    CELESTIAL_SPHERE
} OriginType;

__device__ Real __d;
__device__ Real __camR;
__device__ Real __camTheta;
__device__ Real __camPhi;
__device__ Real __camBeta;
__device__ Real __a;
__device__ Real __a2;
__device__ Real __b1;
__device__ Real __b2;
__device__ Real __ro;
__device__ Real __delta;
__device__ Real __pomega;
__device__ Real __alpha;
__device__ Real __omega;

__device__ Real P(Real r, Real b){
    return r*r + __a2 - __a*b;
}

__device__ Real R(Real r, Parameters param){
    Real b = param.b;
    Real q = param.q;

    Real r2 = r*r;
    Real r4 = r2*r2;
    Real b2 = b*b;

    return r4 - q*r2 - b2*r2 + __a2*r2 + 2*q*r + 2*b2*r - 4*__a*b*r + 2*__a2*r - __a2*q;

}

__device__ Real dbR(Real r, Real b){
    return -2*b*r*r + 4*b*r - 4*__a*r;
}

__device__ Real drR(Real r, Real b, Real q){
    return 4*r*(r*r - __a*b + __a2) - (q + (b-__a)*(b-__a))*(2*r - 2);
}

__device__ Real Theta(Real r, Real theta, Real b, Real q){
    Real sinTheta = sin(theta);
    Real sin2 = sinTheta*sinTheta;

    Real cosTheta = cos(theta);
    Real cos2 = cosTheta*cosTheta;

    return q - cos2*(b*b/sin2 - __a2);
}

__device__ Real dbTheta(Real theta, Real b){
    Real cosTheta = cos(theta);
    Real sinTheta = sin(theta);

    return -(2*b*cosTheta*cosTheta)/(sinTheta*sinTheta);
}

__device__ Real dzTheta(Real theta, Real b){
    Real cosT = cos(theta);
    Real cosT2 = cosT*cosT;

    Real sinT = sin(theta);
    Real sinT2 = sinT*sinT;
    Real sinT3 = sinT2*sinT;
    Real sinT4 = sinT2*sinT2;

    Real b2 = b*b;

    return -2*cosT*(__a2*sinT4 - b2*sinT2 - b2*cosT2)/sinT3;
}

__device__ Real Delta(Real r){
    return r*r - 2*r + __a2;
}

__device__ Real drDelta(Real r){
    return 2*r-2;
}

__device__ Real rho(Real r, Real theta){
    Real cosTheta = cos(theta);
    return sqrt(r*2 + __a2*cosTheta*cosTheta);
}

__device__ Real drRho(Real r, Real theta){
    Real cosT = cos(theta);

    return r/sqrt(__a2*cosT*cosT + r*r);
}

__device__ Real dzRho(Real r, Real theta){
    Real cosT = cos(theta);
    Real sinT = sin(theta);

    return -(__a2*cosT*sinT)/sqrt(__a2*cosT*cosT + r*r);
}

__device__ Real eqMomenta(Parameters param){
    Real r = param.r;
    Real pR = param.pR;
    Real pTheta = param.pTheta;

    Real _Delta = Delta(param.r);
    Real _rho = rho(param.r, param.theta);
    Real tworho2 = 2*_rho*_rho;
    Real _R = R(r, param);
    Real _Theta = Theta(r, param.theta, param.b, param.q);

    Real sol = -(_Delta*pR*pR/tworho2) - (pTheta*pTheta/tworho2) + ((_R+_Delta*_Theta)/(_Delta*tworho2));

    printf("%.10f, %.10f, %.10f, %.10f, %.10f, %.10f, %.10f\n", param.r, param.pR, param.pTheta, param.theta, param.phi, param.b, param.q);

    return sol;
}

__device__ Real eqMomentaTheta(Real theta, Parameters param){
    param.theta = theta;
    return eqMomenta(param);
}

__device__ Real eqMomentaR(Real r, Parameters param){
    param.r = r;
    return eqMomenta(param);
}

__device__ Real eqPhi(Real b, Parameters param){
    Real _R = R(param.r, param);
    Real _Delta = Delta(param.r);
    Real _Theta = Theta(param.r, param.theta, param.b, param.q);
    Real _rho = rho(param.r, param.theta);

    return (_R+_Delta*_Theta)/(2*_Delta*_rho*_rho);
}

__device__ void getCanonicalMomenta(Real rayTheta, Real rayPhi, Real* pR,
                                    Real* pTheta, Real* pPhi){
    // **************************** SET NORMAL **************************** //
    // Cartesian components of the unit vector N pointing in the direction of
    // the incoming ray
    Real Nx = sin(rayTheta) * cos(rayPhi);
    Real Ny = sin(rayTheta) * sin(rayPhi);
    Real Nz = cos(rayTheta);

    // ********************** SET DIRECTION OF MOTION ********************** //
    // Compute denominator, common to all the cartesian components
    Real den = 1. - __camBeta * Ny;

    // Compute factor common to nx and nz
    Real fac = -sqrt(1. - __camBeta*__camBeta);

    // Compute cartesian coordinates of the direction of motion. See(A.9)
    Real nY = (-Ny + __camBeta) / den;
    Real nX = fac * Nx / den;
    Real nZ = fac * Nz / den;

    // Convert the direction of motion to the FIDO's spherical orthonormal
    // basis. See (A.10)
    Real nR = nX;
    Real nTheta = -nZ;
    Real nPhi = nY;

    // *********************** SET CANONICAL MOMENTA *********************** //
    // Compute energy as measured by the FIDO. See (A.11)
    Real E = 1. / (__alpha + __omega * __pomega * nPhi);

    // Set conserved energy to unity. See (A.11)
    // Real pt = -1;

    // Compute the canonical momenta. See (A.11)
    *pR = E * __ro * nR / sqrt(__delta);
    *pTheta = E * __ro * nTheta;
    *pPhi = E * __pomega * nPhi;
}

__device__ void getConservedQuantities(Real pTheta, Real pPhi, Real* b,
                                       Real* q){
    // ********************* GET CONSERVED QUANTITIES ********************* //
    // Get conserved quantities. See (A.12)
    *b = pPhi;
    Real sinTheta = sin(__camTheta);
    *q = pTheta*pTheta + cos(__camTheta)*((*b)*(*b) / sinTheta*sinTheta - __a2);
}

// NOTE: This is b_0(r) - b, not just b0(r)
__device__ Real b0b(Real r, Parameters param){
    Real b = param.b;
    return -((r*r*r - 3.*(r*r) + __a2*r + __a2) / (__a*(r-1.))) - b;
}

__device__ Real q0(Real r){
    Real r3 = r*r*r;
    return -(r3*(r3 - 6.*(r*r) + 9.*r - 4.*__a2)) / (__a2*((r-1.)*(r-1.)));
}

__device__ OriginType getOriginType(Real pR, Real b, Real q){
    Parameters param;

    param.b = b;
    param.q = q;

    // Compute r0 such that b0(r0) = b
    Real r0 = secant(-30., 30., b0b, param);

    OriginType origin;

    if(__b1 < b && b < __b2 && q < q0(r0)){
        if(pR > 0)
            origin = HORIZON;
        else
            origin = CELESTIAL_SPHERE;
    }
    else{
        Real rUp1 = secant(-30., 30., R, param);

        if(__camR < rUp1)
            origin = HORIZON;
        else
            origin = CELESTIAL_SPHERE;
    }

    return origin;
}



/**
 * Computes the value of the threadId-th component of the function
 * F(t) = (f1(t), ..., fn(t)) and stores it in the memory pointed by f
 * @param Real  x  Value of the time in which the system is solved
 * @param Real* y  Initial conditions for the system: a pointer to a vector
 *                 whose lenght shall be the same as the number of equations in
 *                 the system.
 * @param Real* f  Computed value of the function: a pointer to a vector whose
 *                 lenght shall be the same as the number of equations in the
 *                 system.
 */
__device__ void computeComponent(int threadId, Real x, Real* y, Real* f, Real b, Real q){
    Parameters param;

    Real _R, D, Z, rho1, rho2, rho3, rho4;

    switch(threadId) {
            case 0:
                param.r = y[0];
                param.b = b;
                Z = Theta(param.r, param.theta, param.b, param.q);
                break;

            case 1:
                param.theta = y[1];
                param.q = q;
                break;

            case 2:
                param.phi = y[2];
                D = Delta(param.r);
                break;

            case 3:
                param.pR = y[3];
                rho1 = rho(param.r, param.theta);
                rho2 = rho1*rho1;
                rho3 = rho1*rho2;
                rho4 = rho2*rho2;
                break;

            case 4:
                param.pTheta = y[4];
                _R = R(param.r, param);
                break;
    }
    __syncthreads();

    Real dR, dZ, dRho, dD, sum1, sum2, sum3, num, den;

    switch(threadId) {
            case 0:
                f[threadId] = D * param.pR / rho2;
                break;

            case 1:
                f[threadId] = param.pTheta / rho2;
                break;

            case 2:
                dR = dbR(param.r, param.b);
                dZ = dbTheta(param.theta, param.b);

                f[threadId] = - (dR + D*dZ)/(2*D*rho2);
                break;

            case 3:
                dRho = drRho(param.r, param.theta);
                dD = drDelta(param.r);
                dR = drR(param.r, param.b, param.q);

                sum1 = (dRho*2*D - dD*rho1)/(2*rho3);
                sum2 = (dRho*param.pTheta*param.pTheta)/(rho3);

                num = (dR + Z*dD)*D*rho2 - (_R + D*Z)*(dD*D*rho2 + 2*D*rho1*dRho);
                den = 2*D*D*rho4;
                sum3 = num/den;

                f[threadId] = sum1 + sum2 + sum3;
                break;

            case 4:
                dRho = dzRho(param.r, param.theta);
                dZ = dzTheta(param.theta, param.b);

                sum1 = (dRho*D*param.pR*param.pR)/(rho3);
                sum2 = (dRho*param.pTheta*param.pTheta)/(rho3);
                sum3 = (D*(dZ*rho1 - 2*Z*dRho) - 2*_R*dRho)/(2*D*rho3);

                f[threadId] = sum1 + sum2 + sum3;
                break;
    }
}


__global__ void rayTrace(void* devImage, Real imageRows, Real imageCols, Real pixelWidth, Real pixelHeight, Real d, Real camR, Real camTheta, Real camPhi, Real camBeta, Real a, Real b1,Real b2, Real ro, Real delta, Real pomega, Real alpha, Real omega){
    // Shared memory for the initial conditions of this thread
    __shared__ Real initCond[SYSTEM_SIZE];

    // Retrieve the ids of the thread in the block and of the block in the grid
    int threadId = threadIdx.x + threadIdx.y * blockDim.x;
    // int blockId =  blockIdx.x  + blockIdx.y  * gridDim.x;

    if(threadId < SYSTEM_SIZE){
        Real* globalImage = (Real*) devImage;

        // Set global variables, common to all threads and constants
        // Camera constants
        __d = d;
        __camR = camR;
        __camTheta = camTheta;
        __camPhi = camPhi;
        __camBeta = camBeta;

        // Black hole constants
        __a = a;
        __a2 = a*a;
        __b1 = b1;
        __b2 = b2;

        // Kerr constants
        __ro = ro;
        __delta = delta;
        __pomega = pomega;
        __alpha = alpha;
        __omega = omega;

        // Compute pixel position in the physical space
        Real y = - (blockIdx.x - imageCols/2.) * pixelWidth;
        Real z =   (blockIdx.y - imageRows/2.) * pixelHeight;

        // Compute direction of the incoming ray in the camera's reference frame
        Real rayPhi = Pi + atan(y / d);
        Real rayTheta = Pi/2 + atan(z / sqrt(d*d + y*y));

        // Compute canonical momenta of the ray and the conserved quantites b and q
        Real pR, pTheta, pPhi, b, q;
        getCanonicalMomenta(rayTheta, rayPhi, &pR, &pTheta, &pPhi);
        getConservedQuantities(pTheta, pPhi, &b, &q);

        // Check whether the ray comes from the horizon or from the celetial sphere
        int color = getOriginType(pR, b, q);


        __shared__ Real absoluteTol[SYSTEM_SIZE];
        __shared__ Real relativeTol[SYSTEM_SIZE];
        // Populate the initial conditions. Is this parallelization even necessary?
        switch(threadId){
            case 0:
                initCond[0] = __camR;
                relativeTol[0] = 1e-6;
                absoluteTol[0] = 1e-12;
                break;

            case 1:
                initCond[1] = rayTheta;
                relativeTol[1] = 1e-6;
                absoluteTol[1] = 1e-12;
                break;

            case 2:
                initCond[2] = rayPhi;
                relativeTol[2] = 1e-6;
                absoluteTol[2] = 1e-12;
                break;

            case 3:
                initCond[3] = pR;
                relativeTol[3] = 1e-6;
                absoluteTol[3] = 1e-12;
                break;

            case 4:
                initCond[4] = pTheta;
                relativeTol[4] = 1e-6;
                absoluteTol[4] = 1e-12;
                break;
        }
        __syncthreads();

        RK4Solve(computeComponent, b, q,
                 0., -0.1, initCond, -0.01, -0.1,
                 relativeTol, absoluteTol, 0.9, 0.2, 10.0, 0.04, 2.3e-16);


        globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 0] = color;
        globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 1] = color;
        globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 2] = color;
    }
}
