#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>
#include "numericalMethods.cu"

#define SYSTEM_SIZE {{ SYSTEM_SIZE }}
{{ DEBUG }}

#define J I
#define Pi M_PI

#define CELESTIAL_SPHERE 1
#define HORIZON 0

typedef {{ Real }} Real;


__device__ Real __d;
__device__ Real __camR;
__device__ Real __camTheta;
__device__ Real __camPhi;
__device__ Real __camBeta;
__device__ Real __a;
__device__ Real __a2;
__device__ Real __b1;
__device__ Real __b2;
__device__ Real __ro;
__device__ Real __delta;
__device__ Real __pomega;
__device__ Real __alpha;
__device__ Real __omega;

__device__ Real __b;
__device__ Real __q;

__device__ void getCanonicalMomenta(Real rayTheta, Real rayPhi, Real* pR,
                                    Real* pTheta, Real* pPhi){
    // **************************** SET NORMAL **************************** //
    // Cartesian components of the unit vector N pointing in the direction of
    // the incoming ray
    Real Nx = sin(rayTheta) * cos(rayPhi);
    Real Ny = sin(rayTheta) * sin(rayPhi);
    Real Nz = cos(rayTheta);

    // ********************** SET DIRECTION OF MOTION ********************** //
    // Compute denominator, common to all the cartesian components
    Real den = 1. - __camBeta * Ny;

    // Compute factor common to nx and nz
    Real fac = -sqrt(1. - __camBeta*__camBeta);

    // Compute cartesian coordinates of the direction of motion. See(A.9)
    Real nY = (-Ny + __camBeta) / den;
    Real nX = fac * Nx / den;
    Real nZ = fac * Nz / den;

    // Convert the direction of motion to the FIDO's spherical orthonormal
    // basis. See (A.10)
    Real nR = nX;
    Real nTheta = -nZ;
    Real nPhi = nY;

    // *********************** SET CANONICAL MOMENTA *********************** //
    // Compute energy as measured by the FIDO. See (A.11)
    Real E = 1. / (__alpha + __omega * __pomega * nPhi);

    // Set conserved energy to unity. See (A.11)
    Real pt = -1;

    // Compute the canonical momenta. See (A.11)
    *pR = E * __ro * nR / sqrt(__delta);
    *pTheta = E * __ro * nTheta;
    *pPhi = E * __pomega * nPhi;
}

__device__ void getConservedQuantities(Real pTheta, Real pPhi, Real* b,
                                       Real* q){
    // ********************* GET CONSERVED QUANTITIES ********************* //
    // Get conserved quantities. See (A.12)
    *b = pPhi;
    Real sinTheta = sin(__camTheta);
    *q = pTheta*pTheta + cos(__camTheta)*((*b)*(*b) / sinTheta*sinTheta - __a2);
}

// NOTE: This is b_0(r) - b, not just b0(r)
__device__ Real b0b(Real r, Real a, Real b, Real useless){
    Real a2 = a*a;
    return(-((r*r*r - 3.*(r*r) + a2*r + a2) / (a*(r-1.))) - b);
}

__device__ Real q0(Real r, Real a){
    Real r3 = r*r*r;
    Real a2 = a*a;
    return -(r3*(r3 - 6.*(r*r) + 9.*r - 4.*a2)) / (a2*((r-1.)*(r-1.)));
}

__device__ Real R(Real r, Real a, Real b, Real q){
    Real r2 = r*r;
    Real r4 = r2*r2;
    Real a2 = a*a;
    Real b2 = b*b;
    return(r4 -q*r2 - b2*r2 + a2*r2 + 2*q*r + 2*b2*r - 4*a*b*r + 2*a2*r - a2*q);
}

__global__ void rayTrace(void* devImage, Real imageRows, Real imageCols, Real pixelWidth, Real pixelHeight, Real d, Real camR, Real camTheta, Real camPhi, Real camBeta, Real a, Real b1,Real b2, Real ro, Real delta, Real pomega, Real alpha, Real omega){
    // Retrieve the ids of the thread in the block and of the block in the grid
    int threadId = threadIdx.x + threadIdx.y * blockDim.x;
    int blockId =  blockIdx.x  + blockIdx.y  * gridDim.x;

    Real* globalImage = (Real*) devImage;

    // Set global variables
    __d = d;
    __camR = camR;
    __camTheta = camTheta;
    __camPhi = camPhi;
    __camBeta = camBeta;
    __a = a;
    __a2 = a*a;
    __b1 = b1;
    __b2 = b2;
    __ro = ro;
    __delta = delta;
    __pomega = pomega;
    __alpha = alpha;
    __omega = omega;

    // Compute the squares once and for all
    Real a2 = a*a;

    // Compute pixel position in the physical space
    Real y = - (blockIdx.x - imageCols/2.) * pixelWidth;
    Real z =   (blockIdx.y - imageRows/2.) * pixelHeight;

    // Compute direction of the incoming ray in the camera's reference frame
    Real rayPhi = Pi + atan(y / d);
    Real rayTheta = Pi/2 + atan(z / sqrt(d*d + y*y));

    // **************************** SET NORMAL **************************** //
    // Cartesian components of the unit vector N pointing in the direction of
    // the incoming ray
    Real Nx = sin(rayTheta) * cos(rayPhi);
    Real Ny = sin(rayTheta) * sin(rayPhi);
    Real Nz = cos(rayTheta);

    // ********************** SET DIRECTION OF MOTION ********************** //
    // Compute denominator, common to all the cartesian components
    Real den = 1. - camBeta * Ny;

    // Compute factor common to nx and nz
    Real fac = -sqrt(1. - camBeta*camBeta);

    // Compute cartesian coordinates of the direction of motion. See(A.9)
    Real nY = (-Ny + camBeta) / den;
    Real nX = fac * Nx / den;
    Real nZ = fac * Nz / den;

    // Convert the direction of motion to the FIDO's spherical orthonormal
    // basis. See (A.10)
    Real nR = nX;
    Real nTheta = -nZ;
    Real nPhi = nY;

    // *********************** SET CANONICAL MOMENTA *********************** //
    // Compute energy as measured by the FIDO. See (A.11)
    Real E = 1. / (alpha + omega * pomega * nPhi);

    // Set conserved energy to unity. See (A.11)
    Real pt = -1;

    // Compute the canonical momenta. See (A.11)
    Real pR = E * ro * nR / sqrt(delta);
    Real pTheta = E * ro * nTheta;
    Real pPhi = E * pomega * nPhi;

    // ********************* SET CONSERVED QUANTITIES ********************* //
    // Set conserved quantities. See (A.12)
    Real b = pPhi;
    Real sinTheta = sin(camTheta);
    Real q = pTheta*pTheta + cos(camTheta)*(b*b / sinTheta*sinTheta - a2);

    // Real pR, pTheta, pPhi, b, q;
    // getCanonicalMomenta(rayTheta, rayPhi, &pR, &pTheta, &pPhi);
    // getConservedQuantities(rayTheta, rayPhi, &b, &q);

    // Compute r0 such that b0(r0) = b
    Real r0 = secant(-30., 30., b0b, a, b, 0);

    int color = 0.5;

    if(b1 < b && b < b2 && q < q0(r0, a)){
        if(pR > 0)
            color = HORIZON;
        else
            color = CELESTIAL_SPHERE;
    }
    else{
        Real rUp1 = secant(-30., 30., R, a, b, q);

        if(camR < rUp1)
            color = HORIZON;
        else
            color = CELESTIAL_SPHERE;
    }

    globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 0] = color;
    globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 1] = color;
    globalImage[3*(blockIdx.x + blockIdx.y*gridDim.x) + 2] = color;
}





// Compute r0 such that b0(r0) = b. The computation of this number involves
// complex numbers (there is a square root of a negative number).
// Nevertheless, the imaginary parts cancel each other when substracting
// the final terms. In order not to get np.sqrt errors because of the
// negative argument, a conversion to complex is forced summing a null
// imaginary part in the argument of sqrt (see the + 0J below, in the
// innerSqrt assignation). After the final computation is done, the real
// part is retrieved (the imaginary part can be considered null).
//
// // Simplify notation by computing this factor before
// fac = -9. + 3.*a2 + 3.*a*b;
// Real fac3 = fac*fac*fac;
//
// // Compute the square root of a negative number, by creating a complex with
// // real part zero and imaginary part the square root of the absolute value
// // of the number
// Real radicand = (54. - 54.*a2)*(54. - 54.*a2) + 4.*fac3;
//
// if(radicand < 0){
//     Complex innerSqrt = make_hipDoubleComplex(0., sqrt(-radicand));
//     Complex summand = make_hipDoubleComplex(54. - 54.*a2, 0.);
//
//     // Simplify notation by computing this cubic root
//     Complex base = hipCadd(innerSqrt, summand);
//     Complex cubicRoot = cuCpow(base, 1./3.);
//
//     // Finish the computation with the above computed numbers
//     Real cubicTwo = 1.2599210498948732; // pow(2, 1./3.);
//     Complex num1 = make_hipDoubleComplex(cubicTwo*fac, 0.);
//     Complex den1 = make_hipDoubleComplex(3*hipCreal(cubicRoot),
//                                         3*hipCimag(cubicRoot));
//     Complex den2 = make_hipDoubleComplex(3*cubicTwo, 0.);
//
//     Complex one = make_hipDoubleComplex(1., 0.);
//
//     Complex r0_c = hipCsub(one,
//                           hipCadd(hipCdiv(num1,den1),
//                                  hipCdiv(cubicRoot,den2)
//                                 )
//                          );
//
//     // Retrieve the real part and make sure the imaginary part is (nearly) zero
//     Real r0 = hipCreal(r0_c);
//     assert(abs(hipCimag(r0_c)) < 1e-9);
//
//     if(blockIdx.x==170 && blockIdx.y==184){
//         printf("r_0 = %.10f\n", r0);
//     }
// }
